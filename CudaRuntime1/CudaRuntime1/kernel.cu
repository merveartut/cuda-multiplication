﻿
#include "hip/hip_runtime.h"




#include <iostream>




__global__ void matrixMul(const int* a, const int* b, int* c, int size) {
    // Thread'lerin satır ve sütun indexlerini hesapla
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int c_sum = 0;
    // multiplication islemini yap
    c[row * size + col] = 0;
    for (int k = 0; k <size; k++) {
       
       c_sum += a[row * size + k] * b[k * size + col];
    }
    c[row * size + col] = c_sum;   // sonucu c dizisine ata

}

void cpuMul(int* a, int* b, int* c, int N) {
   
    for (int i = 0; i < N; i++) {
        
        for (int j = 0; j < N; j++) {
                                                //her satır ve her sutun icin carpim hesaplama
            int tmp = 0;
            for (int k = 0; k < N; k++) {
               
                tmp += a[i * N + k] * b[k * N + j];
            }

            c[i * N + j] = tmp;
        }
    }
}
int main() {
    
    int n = 1 << 10;

    

    int* h_a;
    int* h_b;
    int* h_c;
    int* h_cc;

    int* d_a;
    int* d_b;
    int* d_c;
    int* d_cc;

    size_t bytes = n * n * sizeof(int);

    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);
    h_cc = (int*)malloc(bytes);

    for (int i = 0; i < n;i++) {           //matrisleri random olustur
        for (int j = 0;j < n;j++) {
            h_a[i * n + j] = rand() % 1024;
            h_b[i * n + j] = rand() % 1024;
        }
    }

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    hipMalloc(&d_cc, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int threads_per_block = 16;
    dim3 blocks(threads_per_block, threads_per_block);
    dim3 grid(n / blocks.x, n / blocks.y);

    matrixMul<<<grid,blocks>>> (d_a, d_b, d_c, n);   //kernel cagrisi gonder

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    cpuMul(h_a, h_b, h_cc, n); //cpu hesaplamasi 

    int ok = 1;
    for (int i = 0; i < n; ++i)        //cpu ve gpu sonuclarini karsilastir
    {
        for (int j = 0; j < n; ++j)
        {
            
            if (h_cc[i * n + j] != h_c[i * n + j])
            {
                ok = 0;
            }
        }
    
    }


    if (ok)
    {
        printf("tum sonuclar dogru!");
    }
    else
    {
        printf("sonuclar yanlis! ");
    }


    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

 
    return 0;
}